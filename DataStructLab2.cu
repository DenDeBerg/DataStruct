#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <mkl.h>
#include <>

// Создание матриц со случайным заполнением числами целого типа
void generateMatrix(float* matrix, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            matrix[i * size + j] = static_cast<float>(rand()) / RAND_MAX;
        }
    }
}

// Вспомогательная функция по перемножениию матриц с использованием CUDA
__global__ void cudaMatrixMultiply(float* A, float* B, float* C, int size) {
    __shared__ float As[16][16];
    __shared__ float Bs[16][16];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = 0;

    float sum = 0.0f;

    for (int ti = threadIdx.x; ti < size; ti += blockDim.x) {
        As[threadIdx.y][threadIdx.x] = A[ti * size + j];
        Bs[threadIdx.y][threadIdx.x] = B[i * size + ti];
        __syncthreads();

        for (int k = 0; k < 16; k++) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (i < size && j < size) {
        C[i * size + j] = sum;
    }
}

// Функция по перемножениию матриц с использованием CUDA
void matrixMultiplyCUDA(float* A, float* B, float* C, int size) {
    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc((void**)&d_A, size * size * sizeof(float));
    hipMalloc((void**)&d_B, size * size * sizeof(float));
    hipMalloc((void**)&d_C, size * size * sizeof(float));

    hipMemcpy(d_A, A, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size * size * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(size / threadsPerBlock.x, size / threadsPerBlock.y);

    cudaMatrixMultiply << <numBlocks, threadsPerBlock >> > (d_A, d_B, d_C, size);

    hipMemcpy(C, d_C, size * size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Функции перемножения матриц с использованием формул из линейной алгебры
void matrixMultiplyNaive(float* A, float* B, float* C, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            for (int k = 0; k < size; k++) {
                C[i * size + j] += A[i * size + k] * B[k * size + j];
            }
        }
    }
}

// Функция перемножения матриц с использованием BLAS
void matrixMultiplyBLAS(float* A, float* B, float* C, int size) {
    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, size, size, size, 1.0f, A, size, B, size, 0.0f, C, size);
}


int main() {
    int size = 1048;
    float* A = new float[size * size];
    float* B = new float[size * size];
    float* C_naive = new float[size * size];
    float* C_blas = new float[size * size];
    float* C_cuda = new float[size * size];

    generateMatrix(A, size);
    generateMatrix(B, size);
    
    matrixMultiplyNaive(A, B, C_naive, size);
    matrixMultiplyBLAS(A, B, C_blas, size);
    matrixMultiplyCUDA(A, B, C_cuda, size);

    clock_t start, end;

    // Метод с использованием формул из линейной алгебры
    start = clock();
    matrixMultiplyNaive(A, B, C_naive, size);
    end = clock();
    double naive_time = static_cast<double>(end - start) / CLOCKS_PER_SEC;
    double naive_mflops = 2.0 * size * size * size / (1024 * 1024 * naive_time);

    // Метод с использованием MKL
    start = clock();
    matrixMultiplyBLAS(A, B, C_blas, size);
    end = clock();
    double blas_time = static_cast<double>(end - start) / CLOCKS_PER_SEC;
    double blas_mflops = 2.0 * size * size * size / (1024 * 1024 * blas_time);

    // Метод с использованием CUDA
    start = clock();
    matrixMultiplyCUDA(A, B, C_cuda, size);
    hipDeviceSynchronize();
    end = clock();
    double cuda_time = static_cast<double>(end - start) / CLOCKS_PER_SEC;
    double cuda_mflops = 2.0 * size * size * size / (1024 * 1024 * cuda_time);

    std::cout << "Performance and time:" << std::endl;
    std::cout << "Naive method: " << naive_mflops << " MFlops, time: " << naive_time << " seconds" << std::endl;
    std::cout << "MKL method: " << blas_mflops << " MFlops, time: " << blas_time << " seconds" << std::endl;
    std::cout << "CUDA method: " << cuda_mflops << " MFlops, time: " << cuda_time << " seconds" << std::endl;

    bool equal = true;
    for (int i = 0; i < size * size; i++) {
        if (C_naive[i] != C_blas[i] || C_naive[i] != C_cuda[i]) {
            equal = false;
            break;
        }
    }
    if (equal) {
        std::cout << "Results are equal" << std::endl;
    }
    else {
        std::cout << "Results are not equal" << std::endl;
    }

    delete[] A;
    delete[] B;
    delete[] C_naive;
    delete[] C_blas;
    delete[] C_cuda;


    return 0;
}